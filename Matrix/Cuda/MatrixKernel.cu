#include "hip/hip_runtime.h"

#include <cmath>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "MatrixKernel.h"

#define TILE_DIM 16


__global__ void cudaMatrixMultiplyKernel(float* a, float* b, float* c, int aRows, int aCols, int bRows, int bCols, int cRows, int cCols)
{
	int row = blockIdx.y * TILE_DIM + threadIdx.y;
	int col = blockIdx.x * TILE_DIM + threadIdx.x;

	float temp = 0;

	for (int i = 0; i < (TILE_DIM + aCols - 1) / TILE_DIM; i++)
		for (int j = 0; j < TILE_DIM; ++j)
			if ((i * TILE_DIM + j < aCols && row < aRows) && (i * TILE_DIM + j < bRows && col < bCols))
				temp += a[row * aCols + i * TILE_DIM + j] * b[(i * TILE_DIM + j) * bCols + col];

	if (row < cRows && col < cCols)
		c[((blockIdx.y * blockDim.y + threadIdx.y) * cCols) + (blockIdx.x * blockDim.x) + threadIdx.x] = temp;

}

void cudaMatrixMultiply(float* a, float* b, float* c, int aRows, int aCols, int bRows, int bCols, int cRows, int cCols)
{
	dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
	dim3 dimGrid;

	dimGrid.x = (cCols + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (cRows + dimBlock.y - 1) / dimBlock.y;

	cudaMatrixMultiplyKernel << <dimGrid, dimBlock >> > (a, b, c, aRows, aCols, bRows, bCols, cRows, cCols);
}